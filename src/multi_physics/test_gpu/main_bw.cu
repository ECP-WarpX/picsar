#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#define PXRMP_GPU __host__ __device__
#define PXRMP_WITH_SI_UNITS
#include "../QED/src/breit_wheeler_engine.hpp"
#include "../QED/src/rng_wrapper.hpp"

//Alias for the picsar::multi_physics namespace
namespace pxrmp =  picsar::multi_physics;

//Seed of the random number generator
const size_t seed = 83871734;

//A lot of particles!
const size_t N = 4000000;

//Sampling parameter for BW pair generation
const size_t sampling = 4;

//How many times should we repeat?
const int repeat = 1;

//Physical constants
const double las_wavlngth = 800.0 * pxrmp::si_nanometer;
const double me_c = pxrmp::electron_mass * pxrmp::light_speed;
const double eref = 2.0*pxrmp::pi*pxrmp::electron_mass*pxrmp::light_speed*pxrmp::light_speed/
            (las_wavlngth*pxrmp::elementary_charge);
const double bref = eref/pxrmp::light_speed;
//____________________________

//Const for momentum initialization
const double mom_coeff = 1000.0*me_c;

//Const for field initialization
const double efl_coeff = 1000.0*eref;
const double bfl_coeff = 1000.0*bref;

//An empty class to be used to call static BW functions
class dummy{};

//Lambda is not needed if SI units are used
const double default_lambda = 1.0;

//timestep to be used in this "simulation"
const double timestep = 1.0e-16;

//Helper function
bool does_file_exist(const char *fileName)
{
    std::ifstream infile(fileName);
    return infile.good();
}



//***********************	BW ENGINE: get_optical_depth	******************************
//GPU kernel to initialize an array of optical depths given an
//array of random numbers [0,1)
__global__
void init_opt_depth(int n, double* opt, double* rand_nums)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n){
		opt[i] = pxrmp::breit_wheeler_engine<double, dummy>::
			internal_get_optical_depth(1.0-rand_nums[i]);
	}
}
//********************************************************************************************



//GPU kernel to initialize fields & momenta randomly
__global__
void init_mom_fields(int n, double* px, double* py, double* pz,
	double*ex, double* ey, double* ez, double* bx, double* by, double* bz, double* ww, double* rand_nums)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n){
		px[i] = mom_coeff*(0.5 - rand_nums[0*n + i]);
		py[i] = mom_coeff*(0.5 - rand_nums[1*n + i]);
		pz[i] = mom_coeff*(0.5 - rand_nums[2*n + i]);
		ex[i] = efl_coeff*(0.5 - rand_nums[3*n + i]);
		ey[i] = efl_coeff*(0.5 - rand_nums[4*n + i]);
		ez[i] = efl_coeff*(0.5 - rand_nums[5*n + i]);
		bx[i] = bfl_coeff*(0.5 - rand_nums[6*n + i]);
		by[i] = bfl_coeff*(0.5 - rand_nums[7*n + i]);
		bz[i] = bfl_coeff*(0.5 - rand_nums[8*n + i]);
		ww[i] = 1.0; // All weights are set to 1
	}
}




//*********************** BW ENGINE: evolve_opt_depth_and_determine_event ******************************
//GPU kernel to test internal_evolve_opt_depth_and_determine_event
__global__
void test_internal_evolve_opt_depth_and_determine_event(
	int n, double* px, double* py, double* pz, double* ex, double* ey, double* ez, double* bx, double* by, double* bz,
	double dt, double* opt, bool* has_event_happened, double* event_dt,
	size_t tab_how_many, double* coords, double* data, pxrmp::breit_wheeler_engine_ctrl<double>* bw_ctrl
)
{
	//Regenerate the lookuptable on GPU
	//This constructor does NOT allocate new memory: it manages existing pointers.
	pxrmp::lookup_1d<double> TTfunctab{tab_how_many, coords, data};

	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n){
		pxrmp::breit_wheeler_engine<double, dummy>::
		internal_evolve_opt_depth_and_determine_event(
         		px[i], py[i], pz[i], ex[i], ey[i], ez[i], bx[i], by[i], bz[i],
         		dt, opt[i], has_event_happened[i], event_dt[i],
         		default_lambda, TTfunctab, *bw_ctrl);

	}
}
//********************************************************************************************


//GPU kernel to set all the has_event_happened to TRUE
__global__
void set_all_events_true(int n, bool* has_event_happened)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
		has_event_happened[i] = true;
}

//*********************** BW ENGINE: generate_breit_wheeler_pairs ******************************
//GPU kernel to test internal_generate_breit_wheeler_pairs
__global__
void test_internal_generate_breit_wheeler_pairs(int n, bool* has_event_happened,
	double* px, double* py, double* pz, double* ex, double* ey, double* ez, double* bx, double* by, double* bz,
	double* weight, size_t sampling,
	double* e_px, double* e_py, double* e_pz, double* p_px, double* p_py, double* p_pz, double* e_weight, double* p_weight,
	size_t tab_how_many_1, double* coords_1, size_t tab_how_many_2, double* coords_2, double* data,
	pxrmp::breit_wheeler_engine_ctrl<double>* bw_ctrl, double* rand_num)
{
	pxrmp::lookup_2d<double> cum_prob_tab{tab_how_many_1, coords_1, tab_how_many_2, coords_2, data};

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int si = sampling*i;

	if (i < n && has_event_happened[i]){
		pxrmp::breit_wheeler_engine<double, dummy>::
		internal_generate_breit_wheeler_pairs(
			px[i], py[i], pz[i], ex[i], ey[i], ez[i], bx[i], by[i], bz[i], weight[i], sampling,
			&e_px[si], &e_py[si], &e_pz[si],
			&p_px[si], &p_py[si], &p_pz[si],
			&e_weight[si], &p_weight[si],
			default_lambda, cum_prob_tab, *bw_ctrl, &rand_num[si]);
	}
}

//********************************************************************************************

int main()
{
	//Seed will be used only with cuRand
	size_t useless_seed = 22051988;

	//Lambda is not used with SI units
	double useless_lambda = 1.0;

	//Change default table parameters in order to speed up the calculations
	//pxrmp::breit_wheeler_engine_ctrl<double> bw_ctrl;
	//bw_ctrl.chi_phot_tdndt_how_many = 200;
	//bw_ctrl.chi_phot_tpair_how_many = 3;
	//bw_ctrl.chi_frac_tpair_how_many = 3;

	//Initialize the BW engine
	auto bw_engine =
		pxrmp::breit_wheeler_engine<double, pxrmp::stl_rng_wrapper<double>>
		{std::move(pxrmp::stl_rng_wrapper<double>{useless_seed}), useless_lambda};//, bw_ctrl};

	//Initialize the lookup tables
   	//Generates tables if they do not exist
   	if(!does_file_exist("tdndt.bin")){
       		bw_engine.compute_dN_dt_lookup_table(&std::cout);
        	bw_engine.write_dN_dt_table("tdndt.bin");
    	}
    	else{
            bw_engine.read_dN_dt_table("tdndt.bin");
    	}

    	if(!does_file_exist("tpair.bin")){
        	bw_engine.compute_cumulative_pair_table(&std::cout);
        	bw_engine.write_cumulative_pair_table("tpair.bin");
    	}
    	else{
        	bw_engine.read_cumulative_pair_table("tpair.bin");
    	}

	//Generates enough random numbers
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	double* d_rand;
	hipMalloc(&d_rand, N*sizeof(double));
	hiprandGenerateUniformDouble(gen, d_rand, N);

	//Allocate on GPU an array of optical depths
	double* d_optical;
	hipMalloc(&d_optical, N*sizeof(double));

	//Initialize the optical depths on GPU
	init_opt_depth<<<(N+255)/256, 256>>>(N, d_optical, d_rand);
	hipDeviceSynchronize();

	//Copy back to the host & print for diag
	double* optical = new double[N];
	hipMemcpy(optical, d_optical, N*sizeof(double), hipMemcpyDeviceToHost);
	std::cout << "Test optical depths: " << std::endl;
	std::cout << optical[0] << " " << optical[1] << " " << optical[N/2] << " " << optical[N-2] << " " << optical[N-1] << std::endl;
	std::cout << "_________" << std::endl << std::endl;

	//Allocate space for momenta & fields & weigths.
	double* d_px;
	double* d_py;
	double* d_pz;
	double* d_ex;
	double* d_ey;
	double* d_ez;
	double* d_bx;
	double* d_by;
	double* d_bz;
	double* d_w;
	hipMalloc(&d_px, N*sizeof(double));
	hipMalloc(&d_py, N*sizeof(double));
	hipMalloc(&d_pz, N*sizeof(double));
	hipMalloc(&d_ex, N*sizeof(double));
	hipMalloc(&d_ey, N*sizeof(double));
	hipMalloc(&d_ez, N*sizeof(double));
	hipMalloc(&d_bx, N*sizeof(double));
	hipMalloc(&d_by, N*sizeof(double));
	hipMalloc(&d_bz, N*sizeof(double));
	hipMalloc(&d_w, N*sizeof(double));

	//Initialize momenta&fields randomly and print a test
	double* d_rand2;
	hipMalloc(&d_rand2, N*sizeof(double)*9);
	hiprandGenerateUniformDouble(gen, d_rand2, N*9);
	init_mom_fields<<<(N+255)/256, 256>>>(N, d_px, d_py, d_pz, d_ex, d_ey, d_ez, d_bx, d_by, d_bz, d_w, d_rand2);
	hipDeviceSynchronize();
	double px, py, pz, ex, ey, ez, bx, by, bz, w;
	hipMemcpy(&px, &d_px[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&py, &d_py[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&pz, &d_pz[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&ex, &d_ex[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&ey, &d_ey[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&ez, &d_ez[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&bx, &d_bx[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&by, &d_by[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&bz, &d_bz[0], sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&w, &d_w[0], sizeof(double), hipMemcpyDeviceToHost);
	std::cout << "Mom & fields & weight : " << std::endl;
	std::vector<double> vv{px,py,pz,ex,ey,ez,bx,by,bz,w};
	for (auto el: vv)
		std::cout << el << " ";
	std::cout << std::endl;
	std::cout << "_________" << std::endl << std::endl;

	//Export innards (in order to copy BW engin data to the GPU)
	auto innards = bw_engine.export_innards();

	//Copy TTfunc_table & bw_ctrl to GPU
	double* d_TTfunc_table_coords;
	double* d_TTfunc_table_data;
	hipMalloc(&d_TTfunc_table_coords, sizeof(double)*innards.TTfunc_table_coords_how_many);
	hipMalloc(&d_TTfunc_table_data, sizeof(double)*innards.TTfunc_table_coords_how_many);
	hipMemcpy(d_TTfunc_table_coords, innards.TTfunc_table_coords_ptr,
		sizeof(double)*innards.TTfunc_table_coords_how_many, hipMemcpyHostToDevice);
	hipMemcpy(d_TTfunc_table_data, innards.TTfunc_table_data_ptr,
		sizeof(double)*innards.TTfunc_table_coords_how_many, hipMemcpyHostToDevice);
	pxrmp::breit_wheeler_engine_ctrl<double>* d_bw_ctrl;
	hipMalloc(&d_bw_ctrl, sizeof(pxrmp::breit_wheeler_engine_ctrl<double>));
	hipMemcpy(d_bw_ctrl, &innards.bw_ctrl, sizeof(pxrmp::breit_wheeler_engine_ctrl<double>), hipMemcpyHostToDevice);

	//Allocate space for has_event_happened and event_dt on GPU
	bool* d_has_event_happened;
	double* d_event_dt;
	hipMalloc(&d_has_event_happened, sizeof(bool)*N);
	hipMalloc(&d_event_dt, sizeof(double)*N);

	//Test internal_evolve_opt_depth_and_determine_event on GPU (multiple times!)
	for(int i = 0; i < repeat; i++){
		test_internal_evolve_opt_depth_and_determine_event<<<(N+255)/256, 256>>>
		(N, d_px, d_py, d_pz, d_ex, d_ey, d_ez, d_bx, d_by, d_bz, timestep, d_optical,
		d_has_event_happened, d_event_dt, innards.TTfunc_table_coords_how_many, d_TTfunc_table_coords,
		d_TTfunc_table_data, d_bw_ctrl);
		hipDeviceSynchronize();
	}

	//Copy optical depths back to the host & print for diag
	double* optical2 = new double[N];
	hipMemcpy(optical2, d_optical, N*sizeof(double), hipMemcpyDeviceToHost);
	std::cout << "Test optical depths: " << std::endl;
	std::cout << optical[0] << " --> " << optical2[0] << std::endl;
	std::cout << optical[1] << " --> " << optical2[1] << std::endl;
	std::cout << optical[N/2] << " --> " << optical2[N/2] << std::endl;
	std::cout << optical[N-2] << " --> " << optical2[N-2] << std::endl;
	std::cout << optical[N-1] << " --> " << optical2[N-1] << std::endl;
	std::cout << "_________" << std::endl << std::endl;

	//For TEST PURPOSES, set all the elements of d_has_event_happened to TRUE
	set_all_events_true<<<(N+255)/256, 256>>>(N, d_has_event_happened);


	//Copy cum_distrib_table from the CPU to the GPU
	double* d_cum_distrib_table_coords_1;
	double* d_cum_distrib_table_coords_2;
	double* d_cum_distrib_table_data;
	hipMalloc(&d_cum_distrib_table_coords_1, sizeof(double)*innards.cum_distrib_table_coords_1_how_many);
	hipMalloc(&d_cum_distrib_table_coords_2, sizeof(double)*innards.cum_distrib_table_coords_2_how_many);
	hipMalloc(&d_cum_distrib_table_data, sizeof(double)*innards.cum_distrib_table_coords_1_how_many*innards.cum_distrib_table_coords_2_how_many);
	hipMemcpy(d_cum_distrib_table_coords_1, innards.cum_distrib_table_coords_1_ptr,
		sizeof(double)*innards.cum_distrib_table_coords_1_how_many,hipMemcpyHostToDevice);
	hipMemcpy(d_cum_distrib_table_coords_2, innards.cum_distrib_table_coords_2_ptr,
		sizeof(double)*innards.cum_distrib_table_coords_2_how_many,hipMemcpyHostToDevice);
	hipMemcpy(d_cum_distrib_table_data, innards.cum_distrib_table_data_ptr,
		sizeof(double)*innards.cum_distrib_table_coords_1_how_many*innards.cum_distrib_table_coords_2_how_many,hipMemcpyHostToDevice);

	//Generate enough random numbers on the GPU
	double* d_rand3;
	hipMalloc(&d_rand3, sizeof(double)*N*sampling);
	hiprandGenerateUniformDouble(gen, d_rand3, N*sampling);

	//Allocate space for momenta & weigths of the generated pairs
	double* d_e_px;
	double* d_e_py;
	double* d_e_pz;
	double* d_p_px;
	double* d_p_py;
	double* d_p_pz;
	double* d_e_w;
	double* d_p_w;
	hipMalloc(&d_e_px, sizeof(double)*N*sampling);
	hipMalloc(&d_e_py, sizeof(double)*N*sampling);
	hipMalloc(&d_e_pz, sizeof(double)*N*sampling);
	hipMalloc(&d_p_px, sizeof(double)*N*sampling);
	hipMalloc(&d_p_py, sizeof(double)*N*sampling);
	hipMalloc(&d_p_pz, sizeof(double)*N*sampling);
	hipMalloc(&d_e_w, sizeof(double)*N*sampling);
	hipMalloc(&d_p_w, sizeof(double)*N*sampling);

	//Test internal_generate_breit_wheeler_pairs on the GPU
	test_internal_generate_breit_wheeler_pairs<<<(N+255)/256, 256>>>
	(N, d_has_event_happened,
	d_px, d_py, d_pz, d_ex, d_ey, d_ez, d_bx, d_by, d_bz,
	d_w, sampling,
	d_e_px, d_e_py, d_e_pz, d_p_px, d_p_py, d_p_pz, d_e_w, d_p_w,
	innards.cum_distrib_table_coords_1_how_many, d_cum_distrib_table_coords_1,
	innards.cum_distrib_table_coords_2_how_many, d_cum_distrib_table_coords_2, d_cum_distrib_table_data,
	d_bw_ctrl,
	d_rand3);


	//Copy some pair properties back to CPU for debug purposes
	double e_px, e_py, e_pz, e_w;
	double p_px, p_py, p_pz, p_w;
	hipMemcpy(&e_px, d_e_px, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&e_py, d_e_py, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&e_pz, d_e_pz, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&e_w, d_e_w, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&p_px, d_p_px, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&p_py, d_p_py, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&p_pz, d_p_pz, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&p_w, d_p_w, sizeof(double), hipMemcpyDeviceToHost);
	std::cout << "Test pairs: " << std::endl;
	std::cout << "e- : " << e_px << " " << e_py << " " << e_pz << " " << e_w << std::endl;
	std::cout << "e+ : " << p_px << " " << p_py << " " << p_pz << " " << p_w << std::endl;
	std::cout << "_________" << std::endl << std::endl;



	//Clean-up
	hipFree(d_optical);
	hipFree(d_rand);
	hipFree(d_px);
	hipFree(d_py);
	hipFree(d_pz);
	hipFree(d_ex);
	hipFree(d_ey);
	hipFree(d_ez);
	hipFree(d_bx);
	hipFree(d_by);
	hipFree(d_bz);
	hipFree(d_w);
	hipFree(d_rand2);
	hipFree(d_TTfunc_table_coords);
	hipFree(d_TTfunc_table_data);
	hipFree(d_bw_ctrl);
    hipFree(d_cum_distrib_table_coords_1);
    hipFree(d_cum_distrib_table_coords_2);
    hipFree(d_cum_distrib_table_data);
	hipFree(d_has_event_happened);
	hipFree(d_event_dt);
	hipFree(d_rand3);
	hipFree(d_e_px);
	hipFree(d_e_py);
	hipFree(d_e_pz);
	hipFree(d_p_px);
	hipFree(d_p_py);
	hipFree(d_p_pz);
	hipFree(d_e_w);
	hipFree(d_p_w);
	delete[] optical;
	delete[] optical2;

	return 0;
}
